

#include <hip/hip_runtime.h>
#include <type_traits>

#ifdef _WIN32
#define EXPORT __declspec(dllexport)
#else
#define EXPORT
#endif

using tt = std::true_type;
using ft = std::false_type;
EXPORT int __host__ shared_cuda11_func(int x)
{
  return x * x + std::integral_constant<int, 17>::value;
}
